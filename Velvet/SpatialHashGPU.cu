#include "hip/hip_runtime.h"
#include "SpatialHashGPU.cuh"

#include <cub/device/device_radix_sort.cuh>

#include "Timer.hpp"
#include "VtBuffer.hpp"

using namespace Velvet;

__device__ __constant__ float d_hashCellSpacing;
__device__ __constant__ int d_hashTableSize;
int h_hashTableSize;

__device__ inline int ComputeIntCoord(float value)
{
	return (int)floor(value / d_hashCellSpacing);
}

__device__ inline int HashCoords(int x, int y, int z)
{
	int h = (x * 92837111) ^ (y * 689287499) ^ (z * 283923481);	// fantasy function
	return abs(h % d_hashTableSize);
}

__device__ inline int HashPosition(glm::vec3 position)
{
	int x = ComputeIntCoord(position.x);
	int y = ComputeIntCoord(position.y);
	int z = ComputeIntCoord(position.z);

	int h = HashCoords(x, y, z);
	return h;
}

// TODO(low): make all parameters conform (output, input, constants)
__global__ void ComputeParticleHash(
	uint* particleHash,
	uint* particleIndex,
	CONST(glm::vec3*) positions,
	uint numObjects)
{
	GET_CUDA_ID(id, numObjects);
	particleHash[id] = HashPosition(positions[id]);
	particleIndex[id] = id;
}

__global__ void FindCellStart(
	uint* cellStart,
	uint* cellEnd,
	CONST(uint*) particleHash,
	uint numObjects)
{
	extern __shared__ uint sharedHash[];

	GET_CUDA_ID_NO_RETURN(id, numObjects);

	uint hash = particleHash[id];
	sharedHash[threadIdx.x + 1] = hash;
	if (id > 0 && threadIdx.x == 0)
	{
		sharedHash[0] = particleHash[id - 1];
	}
	__syncthreads();

	if (id >= numObjects) return;

	if (id == 0 || hash != sharedHash[threadIdx.x])
	{
		cellStart[hash] = id;

		if (id > 0)
		{
			cellEnd[sharedHash[threadIdx.x]] = id;
		}
	}

	if (id == numObjects - 1)
	{
		cellEnd[hash] = id + 1;
	}
}


__global__ void CacheNeighbors_BF(
	uint* neighbors,
	CONST(uint*) particleIndex,
	CONST(uint*) cellStart,
	CONST(glm::vec3*) positions,
	const uint numObjects,
	const uint maxNumNeihgbors)
{
	GET_CUDA_ID(id, numObjects);
	int neighborIndex = id * maxNumNeihgbors;
	for (int neighbor = 0; neighbor < numObjects; neighbor++)
	{
		float distance = glm::length(positions[id] - positions[neighbor]);
		if (neighbor != id && distance < d_hashCellSpacing)
		{
			neighbors[neighborIndex++] = neighbor;
		}
	}
}

__global__ void CacheNeighbors(
	uint* neighbors,
	CONST(uint*) particleIndex,
	CONST(uint*) cellStart,
	CONST(uint*) cellEnd,
	CONST(glm::vec3*) positions,
	CONST(glm::vec3*) originalPositions,
	const uint numObjects,
	const uint maxNumNeihgbors)
{
	GET_CUDA_ID(id, numObjects);

	glm::vec3 position = positions[id];
	glm::vec3 originalPos = originalPositions[id];
	int ix = ComputeIntCoord(position.x);
	int iy = ComputeIntCoord(position.y);
	int iz = ComputeIntCoord(position.z);

	int neighborIndex = id * maxNumNeihgbors;
	for (int x = ix - 1; x <= ix + 1; x++)
	{
		for (int y = iy - 1; y <= iy + 1; y++)
		{
			for (int z = iz - 1; z <= iz + 1; z++)
			{
				int h = HashCoords(x, y, z);
				int start = cellStart[h];
				if (start == 0xffffffff) continue;

				int end = min(cellEnd[h], start+ maxNumNeihgbors);

				for (int i = start; i < end; i++)
				{
					uint neighbor = particleIndex[i];
					float distance = glm::length(position - positions[neighbor]);
					// ignore collision when particles are initially close
					bool filterCollision = glm::length(originalPos - originalPositions[neighbor]) > d_hashCellSpacing;
					if (distance < d_hashCellSpacing && filterCollision)
					{
						neighbors[neighborIndex++] = neighbor;
					}
				}
			}
		}
	}
	if (neighborIndex < (id+1) * maxNumNeihgbors)
	{
		neighbors[neighborIndex] = 0xffffffff;
	}
}

// Cub::sort provides better performance (2x) than thrust
void Sort(
	uint* d_keys_in,
	uint* d_values_in,
	int num_items)
{
	//static void* d_temp_storage = NULL;
	static VtBuffer<void*> d_temp_storage;
	static size_t temp_storage_bytes = 0;


	int maxBit = (int)ceil(log2(h_hashTableSize));
	// Determine temporary device storage requirements
	size_t new_storage_bytes = 0;
	hipcub::DeviceRadixSort::SortPairs(NULL, new_storage_bytes,
		d_keys_in, d_keys_in, d_values_in, d_values_in, num_items);

	if (temp_storage_bytes != new_storage_bytes)
	{
		temp_storage_bytes = new_storage_bytes;
		d_temp_storage.resize(temp_storage_bytes);
	}

	// Run sorting operation
	hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
		d_keys_in, d_keys_in, d_values_in, d_values_in, num_items, 0, maxBit);
}

void Velvet::HashObjects(
	uint* particleHash,
	uint* particleIndex,
	uint* cellStart,
	uint* cellEnd,
	uint* neighbors,
	CONST(glm::vec3*) positions,
	CONST(glm::vec3*) originalPositions,
	const uint numObjects,
	const uint maxNumNeighbors,
	const float hashCellSpacing, 
	const int hashTableSize)
{
	{
		ScopedTimerGPU timer("Solver_HashParticle");

		h_hashTableSize = hashTableSize;
		checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_hashCellSpacing), &hashCellSpacing, sizeof(float)));
		checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_hashTableSize), &hashTableSize, sizeof(int)));
		CUDA_CALL(ComputeParticleHash, numObjects)(particleHash, particleIndex, positions, numObjects);
	}

	{
		ScopedTimerGPU timer("Solver_HashSort");
		Sort(particleHash, particleIndex, numObjects);
	}

	{
		ScopedTimerGPU timer("Solver_HashBuildCell");
		hipMemsetAsync(cellStart, 0xffffffff, sizeof(uint) * (hashTableSize + 1));
		uint numBlocks, numThreads;
		ComputeGridSize(numObjects, numBlocks, numThreads);
		uint smemSize = sizeof(uint) * (numThreads + 1);
		CUDA_CALL_V(FindCellStart, numBlocks, numThreads, smemSize)(cellStart, cellEnd, particleHash, numObjects);
	}
	{
		ScopedTimerGPU timer("Solver_HashCache");
		CUDA_CALL(CacheNeighbors, numObjects)(neighbors, particleIndex, cellStart, cellEnd, positions, originalPositions, numObjects, maxNumNeighbors);
	}
}

