#include "hip/hip_runtime.h"
#include "VtClothSolverGPU.cuh"

#include <tuple>
#include <fmt/format.h>

#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>

using namespace std;

#define EPSILON 1e-6f
#define GET_CUDA_ID(id, maxID) 	uint id = blockIdx.x * blockDim.x + threadIdx.x; if (id >= maxID) return

namespace Velvet
{
	__device__ __constant__ SimulationParams d_params;
	SimulationParams h_params;

	void SetSimulationParams(SimulationParams* hostParams)
	{
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_params), hostParams, sizeof(SimulationParams)));
		h_params = *hostParams;
	}

	const uint blockSize = 256;

	void ComputeGridSize(uint n, uint& numBlocks, uint& numThreads)
	{
		if (n == 0)
		{
			//fmt::print("Error(Solver): numParticles is 0\n");
			numBlocks = 0;
			numThreads = 0;
			return;
		}
		numThreads = min(n, blockSize);
		numBlocks = (n % numThreads != 0) ? (n / numThreads + 1) : (n / numThreads);
	}

	struct InitializePositionsFunctor
	{
		const glm::mat4 matrix;
		InitializePositionsFunctor(glm::mat4 _matrix) : matrix(_matrix) {}

		__host__ __device__
			glm::vec3 operator()(const glm::vec3 position) const {
			return glm::vec3(matrix * glm::vec4(position, 1));
		}
	};

	void InitializePositions(glm::vec3* positions, int count, glm::mat4 modelMatrix)
	{
		thrust::device_ptr<glm::vec3> d_positions(positions);
		thrust::transform(d_positions, d_positions + count, d_positions, InitializePositionsFunctor(modelMatrix));
	}

	__global__ void EstimatePositions_Impl(READ_ONLY(glm::vec3*) positions, glm::vec3* predicted, glm::vec3* velocities, float deltaTime)
	{
		GET_CUDA_ID(id, d_params.numParticles);

		glm::vec3 gravity = glm::vec3(0, -10, 0);
		velocities[id] += d_params.gravity * deltaTime;
		predicted[id] = positions[id] + velocities[id] * deltaTime;
	}

	void EstimatePositions(READ_ONLY(glm::vec3*) positions, glm::vec3* predicted, glm::vec3* velocities, float deltaTime)
	{
		uint numBlocks, numThreads;
		ComputeGridSize(h_params.numParticles, numBlocks, numThreads);
		EstimatePositions_Impl <<< numBlocks, numThreads >>> (positions, predicted, velocities, deltaTime);
	}

	__device__ void AtomicAdd(glm::vec3* address, int index, glm::vec3 val)
	{
		atomicAdd(&(address[index].x), val.x);
		atomicAdd(&(address[index].y), val.y);
		atomicAdd(&(address[index].z), val.z);
	}

	__global__ void SolveStretch_Impl(glm::vec3* predicted, int* stretchIndices, float* stretchLengths, float* inverseMass, uint numConstraints)
	{
		GET_CUDA_ID(id, numConstraints);

		int idx1 = stretchIndices[2 * id];
		int idx2 = stretchIndices[2 * id + 1];
		float expectedDistance = stretchLengths[id];

		glm::vec3 diff = predicted[idx1] - predicted[idx2];
		float distance = glm::length(diff);
		float w1 = inverseMass[idx1];
		float w2 = inverseMass[idx2];

		if (distance > expectedDistance && w1 + w2 > 0)
		{
			auto gradient = diff / (distance + EPSILON);
			// compliance is zero, therefore XPBD=PBD
			auto denom = w1 + w2;
			auto lambda = (distance - expectedDistance) / denom;
			auto correction1 = -w1 * lambda * gradient;
			auto correction2 = w2 * lambda * gradient;
			AtomicAdd(predicted, idx1, correction1);
			AtomicAdd(predicted, idx2, correction2);
		}
	}

	void SolveStretch(glm::vec3* predicted, int* stretchIndices, float* stretchLengths, float* inverseMass, uint numConstraints)
	{
		uint numBlocks, numThreads;
		ComputeGridSize(numConstraints, numBlocks, numThreads);
		SolveStretch_Impl <<< numBlocks, numThreads >>> (predicted, stretchIndices, stretchLengths, inverseMass, numConstraints);
	}

	__global__ void UpdatePositionsAndVelocities_Impl(READ_ONLY(glm::vec3*) predicted, glm::vec3* velocities, glm::vec3* positions, float deltaTime)
	{
		// TODO: encapsulate macro
		uint id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= d_params.numParticles) return;

		velocities[id] = (predicted[id] - positions[id]) / deltaTime;// * (1 - d_params.damping * deltaTime);
		positions[id] = predicted[id];
	}

	void UpdatePositionsAndVelocities(READ_ONLY(glm::vec3*) predicted, glm::vec3* velocities, glm::vec3* positions, float deltaTime)
	{
		uint numBlocks, numThreads;
		ComputeGridSize(h_params.numParticles, numBlocks, numThreads);
		UpdatePositionsAndVelocities_Impl <<< numBlocks, numThreads >>> (predicted, velocities, positions, deltaTime);
	}

	__global__ void SolveAttachment_Impl(int numConstraints, READ_ONLY(int*) attachIndices, READ_ONLY(glm::vec3*) attachPositions, glm::vec3* predicted)
	{
		GET_CUDA_ID(id, numConstraints);

		predicted[attachIndices[id]] = attachPositions[id];
	}

	void SolveAttachment(int numConstraints, READ_ONLY(int*) attachIndices, READ_ONLY(glm::vec3*) attachPositions, glm::vec3* predicted)
	{
		uint numBlocks, numThreads;
		ComputeGridSize(numConstraints, numBlocks, numThreads);
		SolveAttachment_Impl <<<numBlocks, numThreads >>> (numConstraints, attachIndices, attachPositions, predicted);
	}
}