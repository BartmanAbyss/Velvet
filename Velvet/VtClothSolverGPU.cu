#include "hip/hip_runtime.h"
#include "VtClothSolverGPU.cuh"
#include "Common.hpp"
#include "Common.cuh"
#include "Timer.hpp"

using namespace std;

namespace Velvet
{
	__device__ __constant__ VtSimParams d_params;
	VtSimParams h_params;

	void SetSimulationParams(VtSimParams* hostParams)
	{
		ScopedTimerGPU timer("Solver_SetParams");
		checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_params), hostParams, sizeof(VtSimParams)));
		h_params = *hostParams;
	}

	__global__ void InitializePositions_Impl(glm::vec3* positions, const int count, const glm::mat4 modelMatrix)
	{
		GET_CUDA_ID(id, count);
		positions[id] = modelMatrix * glm::vec4(positions[id], 1);
	}

	void InitializePositions(glm::vec3* positions, int count, glm::mat4 modelMatrix)
	{
		ScopedTimerGPU timer("Solver_Initialize");
		CUDA_CALL(InitializePositions_Impl, count)(positions, count, modelMatrix);
	}

	__global__ void PredictPositions_Impl(CONST(glm::vec3*) positions, glm::vec3* predicted, glm::vec3* velocities, float deltaTime)
	{
		GET_CUDA_ID(id, d_params.numParticles);

		glm::vec3 gravity = glm::vec3(0, -10, 0);
		velocities[id] += d_params.gravity * deltaTime;
		predicted[id] = positions[id] + velocities[id] * deltaTime;
	}

	void PredictPositions(CONST(glm::vec3*) positions, glm::vec3* predicted, glm::vec3* velocities, float deltaTime)
	{
		ScopedTimerGPU timer("Solver_Predict");
		CUDA_CALL(PredictPositions_Impl, h_params.numParticles)(positions, predicted, velocities, deltaTime);
	}

	__device__ void AtomicAdd(glm::vec3* address, int index, glm::vec3 val, int reorder)
	{
		int r1 = reorder % 3;
		int r2 = (reorder+1) % 3;
		int r3 = (reorder+2) % 3;
		atomicAdd(&(address[index].x)+r1, val[r1]);
		atomicAdd(&(address[index].x)+r2, val[r2]);
		atomicAdd(&(address[index].x)+r3, val[r3]);
	}

	__global__ void SolveStretch_Impl(uint numConstraints, CONST(int*) stretchIndices, CONST(float*) stretchLengths, 
		CONST(float*) inverseMass, CONST(glm::vec3*) predicted, glm::vec3* positionDeltas, int* positionDeltaCount)
	{
		GET_CUDA_ID(id, numConstraints);

		int idx1 = stretchIndices[2 * id];
		int idx2 = stretchIndices[2 * id + 1];
		float expectedDistance = stretchLengths[id];

		glm::vec3 diff = predicted[idx1] - predicted[idx2];
		float distance = glm::length(diff);
		float w1 = inverseMass[idx1];
		float w2 = inverseMass[idx2];

		if (distance != expectedDistance && w1 + w2 > 0)
		{
			glm::vec3 gradient = diff / (distance + EPSILON);
			// compliance is zero, therefore XPBD=PBD
			float denom = w1 + w2;
			float lambda = (distance - expectedDistance) / denom;
			glm::vec3 common = lambda * gradient;
			glm::vec3 correction1 = -w1 * common;
			glm::vec3 correction2 = w2 * common;
			int reorder = idx1 + idx2;
			AtomicAdd(positionDeltas, idx1, correction1, reorder);
			AtomicAdd(positionDeltas, idx2, correction2, reorder);
			atomicAdd(&positionDeltaCount[idx1], 1);
			atomicAdd(&positionDeltaCount[idx2], 1);
			//printf("correction[%d] = (%.2f,%.2f,%.2f)\n", idx1, correction1.x, correction1.y, correction1.z);
			//printf("correction[%d] = (%.2f,%.2f,%.2f)\n", idx2, correction2.x, correction2.y, correction2.z);
		}
	}


	void SolveStretch(uint numConstraints, CONST(int*) stretchIndices, CONST(float*) stretchLengths,
		CONST(float*) inverseMass, glm::vec3* predicted, glm::vec3* positionDeltas, int* positionDeltaCount)
	{
		ScopedTimerGPU timer("Solver_SolveStretch");
		CUDA_CALL(SolveStretch_Impl, numConstraints)(numConstraints, stretchIndices, stretchLengths, inverseMass, predicted, positionDeltas, positionDeltaCount);
	}

	__global__ void SolveBending_Impl(
		glm::vec3* predicted,
		glm::vec3* positionDeltas,
		int* positionDeltaCount,
		CONST(uint*) bendingIndices,
		CONST(float*) bendingAngles,
		CONST(float*) invMass,
		uint numConstraints,
		float deltaTime)
	{
		GET_CUDA_ID(id, numConstraints);
		uint idx1 = bendingIndices[id * 4];
		uint idx2 = bendingIndices[id * 4+1];
		uint idx3 = bendingIndices[id * 4+2];
		uint idx4 = bendingIndices[id * 4+3];
		float expectedAngle = bendingAngles[id];

		float w1 = invMass[idx1];
		float w2 = invMass[idx2];
		float w3 = invMass[idx3];
		float w4 = invMass[idx4];

		glm::vec3 p1 = predicted[idx1];
		glm::vec3 p2 = predicted[idx2] - p1;
		glm::vec3 p3 = predicted[idx3] - p1;
		glm::vec3 p4 = predicted[idx4] - p1;
		glm::vec3 n1 = glm::normalize(glm::cross(p2, p3));
		glm::vec3 n2 = glm::normalize(glm::cross(p2, p4));

		float d = clamp(glm::dot(n1, n2), 0.0f, 1.0f);
		float angle = acos(d);
		// cross product for two equal vector produces NAN
		if (angle < EPSILON || isnan(d)) return;

		glm::vec3 q3 = (glm::cross(p2, n2) + glm::cross(n1, p2) * d) / (glm::length(glm::cross(p2, p3)) + EPSILON);
		glm::vec3 q4 = (glm::cross(p2, n1) + glm::cross(n2, p2) * d) / (glm::length(glm::cross(p2, p4)) + EPSILON);
		glm::vec3 q2 = -(glm::cross(p3, n2) + glm::cross(n1, p3) * d) / (glm::length(glm::cross(p2, p3)) + EPSILON)
			- (glm::cross(p4, n1) + glm::cross(n2, p4) * d) / (glm::length(glm::cross(p2, p4)) + EPSILON);
		glm::vec3 q1 = -q2 - q3 - q4;

		float xpbd_bend = d_params.bendCompliance / deltaTime / deltaTime;
		float denom = xpbd_bend + (w1 * glm::dot(q1, q1) + w2 * glm::dot(q2, q2) + w3 * glm::dot(q3, q3) + w4 * glm::dot(q4, q4));
		if (denom < EPSILON) return; // ?
		float lambda = sqrt(1.0f - d * d) * (angle - expectedAngle) / denom;

		int reorder = idx1 + idx2 + idx3 + idx4;
		AtomicAdd(positionDeltas, idx1, w1 * lambda * q1, reorder);
		AtomicAdd(positionDeltas, idx2, w2 * lambda * q2, reorder);
		AtomicAdd(positionDeltas, idx3, w3 * lambda * q3, reorder);
		AtomicAdd(positionDeltas, idx4, w4 * lambda * q4, reorder);
		
		atomicAdd(&positionDeltaCount[idx1], 1);
		atomicAdd(&positionDeltaCount[idx2], 1);
		atomicAdd(&positionDeltaCount[idx3], 1);
		atomicAdd(&positionDeltaCount[idx4], 1);
	}

	void SolveBending(
		glm::vec3* predicted,
		glm::vec3* positionDeltas,
		int* positionDeltaCount,
		CONST(uint*) bendingIndices,
		CONST(float*) bendingAngles,
		CONST(float*) invMass,
		uint numConstraints,
		float deltaTime)
	{
		ScopedTimerGPU timer("Solver_SolveBending");
		CUDA_CALL(SolveBending_Impl, numConstraints)(predicted, positionDeltas, positionDeltaCount, bendingIndices, bendingAngles, invMass, numConstraints, deltaTime);
	}

	__global__ void SolveAttachment_Impl(
		int numConstraints,
		CONST(float*) invMass,
		CONST(int*) attachIndices, 
		CONST(glm::vec3*) attachPositions, 
		CONST(float*) attachDistances,
		CONST(glm::vec3*) predicted,
		glm::vec3* positionDeltas,
		int* positionDeltaCount)
	{
		GET_CUDA_ID(id, numConstraints);

		uint pid = attachIndices[id];

		glm::vec3 attachPoint = attachPositions[id];
		float targetDist = attachDistances[id] * d_params.longRangeStretchiness;
		if (invMass[pid] == 0 && targetDist > 0) return;

		glm::vec3 pred = predicted[pid];
		glm::vec3 diff = pred - attachPoint;
		float dist = glm::length(diff);

		if (dist > targetDist)
		{
			//float coefficient = max(targetDist, dist - 0.1*d_params.particleDiameter);// 0.05 * targetDist + 0.95 * dist;
			glm::vec3 correction = -diff + diff / dist * targetDist;
			AtomicAdd(positionDeltas, pid, correction, id);
			atomicAdd(&positionDeltaCount[pid], 1);
		}
	}

	void SolveAttachment(
		int numConstraints,
		CONST(float*) invMass,
		CONST(int*) attachIndices,
		CONST(glm::vec3*) attachPositions,
		CONST(float*) attachDistances,
		glm::vec3* predicted,
		glm::vec3* positionDeltas,
		int* positionDeltaCount)
	{
		ScopedTimerGPU timer("Solver_SolveAttach");
		CUDA_CALL(SolveAttachment_Impl, numConstraints)(numConstraints, invMass, attachIndices, attachPositions, attachDistances, predicted, positionDeltas, positionDeltaCount);
	}

	__global__ void ApplyDeltas_Impl(glm::vec3* predicted, glm::vec3* positionDeltas, int* positionDeltaCount)
	{
		GET_CUDA_ID(id, d_params.numParticles);

		float count = (float)positionDeltaCount[id];
		if (count > 0)
		{
			predicted[id] += positionDeltas[id] / count * d_params.relaxationFactor;
			positionDeltas[id] = glm::vec3(0);
			positionDeltaCount[id] = 0;
		}
	}

	void ApplyDeltas(glm::vec3* predicted, glm::vec3* positionDeltas, int* positionDeltaCount)
	{
		ScopedTimerGPU timer("Solver_ApplyDeltas");
		CUDA_CALL(ApplyDeltas_Impl, h_params.numParticles)(predicted, positionDeltas, positionDeltaCount);
	}

	__device__ glm::vec3 ComputeFriction(glm::vec3 correction, glm::vec3 relVel)
	{
		glm::vec3 friction = glm::vec3(0);
		float correctionLength = glm::length(correction);
		if (d_params.friction > 0 && correctionLength > 0)
		{
			glm::vec3 norm = correction / correctionLength;

			glm::vec3 tanVel = relVel - norm * glm::dot(relVel, norm);
			float tanLength = glm::length(tanVel);
			float maxTanLength = correctionLength * d_params.friction;

			friction = -tanVel * min(maxTanLength / tanLength, 1.0f);
		}
		return friction;
	}

	__global__ void CollideSDF_Impl(const uint numColliders, CONST(SDFCollider*) colliders, CONST(glm::vec3*) positions, glm::vec3* predicted)
	{
		GET_CUDA_ID(id, d_params.numParticles);

		auto pos = positions[id];
		auto pred = predicted[id];
		for (int i = 0; i < numColliders; i++)
		{
			auto collider = colliders[i];
			glm::vec3 correction = collider.ComputeSDF(pred, d_params.collisionMargin);
			pred += correction;

			glm::vec3 relVel = pred - pos;
			auto friction = ComputeFriction(correction, relVel);
			pred += friction;
		}
		predicted[id] = pred;
	}

	void CollideSDF(const uint numColliders, CONST(SDFCollider*) colliders, CONST(glm::vec3*) positions, glm::vec3* predicted)
	{
		ScopedTimerGPU timer("Solver_CollideSDFs");
		if (numColliders == 0) return;
		
		CUDA_CALL(CollideSDF_Impl, h_params.numParticles)(numColliders, colliders, positions, predicted);
	}

	__global__ void CollideParticles_Impl(
		CONST(float*) inverseMass,
		CONST(uint*) neighbors,
		CONST(glm::vec3*) positions,
		glm::vec3* predicted)
	{
		GET_CUDA_ID(id, d_params.numParticles);

		glm::vec3 positionDelta = glm::vec3(0);
		int deltaCount = 0;
		glm::vec3 pred_i = predicted[id];
		glm::vec3 vel_i = (pred_i - positions[id]);
		float w_i = inverseMass[id];

		for (int neighbor = id * d_params.maxNumNeighbors; neighbor < (id + 1) * d_params.maxNumNeighbors; neighbor++)
		{
			uint j = neighbors[neighbor];
			if (j > d_params.numParticles) break;
			//if (j > id) continue;

			float expectedDistance = d_params.particleDiameter;

			glm::vec3 pred_j = predicted[j];
			glm::vec3 diff = pred_i - pred_j;
			float distance = glm::length(diff);
			float w_j = inverseMass[j];

			if (distance < expectedDistance && w_i + w_j > 0)
			{
				glm::vec3 gradient = diff / (distance + EPSILON);
				float denom = w_i + w_j;
				float lambda = (distance - expectedDistance) / denom;
				glm::vec3 common = lambda * gradient;

				positionDelta -= w_i * common;

				glm::vec3 relativeVelocity = vel_i - (pred_j - positions[j]);
				glm::vec3 friction = ComputeFriction(common, relativeVelocity);
				positionDelta += w_i * friction;
			}
		}

		predicted[id] += positionDelta;
	}

	void CollideParticles(
		CONST(float*) inverseMass,
		CONST(uint*) neighbors,
		CONST(glm::vec3*) positions,
		glm::vec3* predicted)
	{
		ScopedTimerGPU timer("Solver_CollideParticles");
		CUDA_CALL(CollideParticles_Impl, h_params.numParticles)(inverseMass, neighbors, positions, predicted);
	}

	__global__ void Finalize_Impl(CONST(glm::vec3*) predicted, glm::vec3* velocities, glm::vec3* positions, float deltaTime)
	{
		GET_CUDA_ID(id, d_params.numParticles);

		glm::vec3 raw_vel = (predicted[id] - positions[id]) / deltaTime;
		float raw_vel_len = glm::length(raw_vel);
		glm::vec3 new_pos = predicted[id];
		if (raw_vel_len > d_params.maxSpeed)
		{
			raw_vel = raw_vel / raw_vel_len * d_params.maxSpeed;
			new_pos = positions[id] + raw_vel * deltaTime;
			//printf("Limit vel[%.3f>%.3f] for id[%d]. Pred[%.3f,%.3f,%.3f], Pos[%.3f,%.3f,%.3f]\n", raw_vel_len, d_params.maxSpeed, id);
		}
		velocities[id] = raw_vel * (1 - d_params.damping * deltaTime);
		positions[id] = new_pos;
	}

	void Finalize(CONST(glm::vec3*) predicted, glm::vec3* velocities, glm::vec3* positions, float deltaTime)
	{
		ScopedTimerGPU timer("Solver_Finalize");
		CUDA_CALL(Finalize_Impl, h_params.numParticles)(predicted, velocities, positions, deltaTime);
	}

	__global__ void ComputeTriangleNormals(uint numTriangles, CONST(glm::vec3*) positions, CONST(uint*) indices, glm::vec3* normals)
	{
		GET_CUDA_ID(id, numTriangles);
		uint idx1 = indices[id * 3];
		uint idx2 = indices[id * 3+1];
		uint idx3 = indices[id * 3+2];

		auto p1 = positions[idx1];
		auto p2 = positions[idx2];
		auto p3 = positions[idx3];

		auto normal = glm::cross(p2 - p1, p3 - p1);
		int reorder = idx1 + idx2 + idx3;
		AtomicAdd(normals, idx1, normal, reorder);
		AtomicAdd(normals, idx2, normal, reorder);
		AtomicAdd(normals, idx3, normal, reorder);
	}

	__global__ void ComputeVertexNormals(glm::vec3* normals)
	{
		GET_CUDA_ID(id, d_params.numParticles);

		//normals[id] = glm::vec3(0,1,0);
		normals[id] = glm::normalize(normals[id]);
	}

	void ComputeNormal(uint numTriangles, CONST(glm::vec3*) positions, CONST(uint*) indices, glm::vec3* normals)
	{
		ScopedTimerGPU timer("Solver_UpdateNormals");
		if (h_params.numParticles)
		{
			hipMemsetAsync(normals, 0, h_params.numParticles * sizeof(glm::vec3));
			CUDA_CALL(ComputeTriangleNormals, numTriangles)(numTriangles, positions, indices, normals);
			CUDA_CALL(ComputeVertexNormals, h_params.numParticles)(normals);
		}
	}

}