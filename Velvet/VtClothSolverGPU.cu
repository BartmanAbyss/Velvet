#include "hip/hip_runtime.h"
#include "VtClothSolverGPU.cuh"
#include "Common.hpp"
#include "Common.cuh"

using namespace std;

namespace Velvet
{
	__device__ __constant__ VtSimParams d_params;
	VtSimParams h_params;

	void SetSimulationParams(VtSimParams* hostParams)
	{
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_params), hostParams, sizeof(VtSimParams)));
		h_params = *hostParams;
	}

	struct InitializePositionsFunctor
	{
		const glm::mat4 matrix;
		InitializePositionsFunctor(glm::mat4 _matrix) : matrix(_matrix) {}

		__host__ __device__
			glm::vec3 operator()(const glm::vec3 position) const {
			return glm::vec3(matrix * glm::vec4(position, 1));
		}
	};

	void InitializePositions(glm::vec3* positions, int count, glm::mat4 modelMatrix)
	{
		thrust::device_ptr<glm::vec3> d_positions(positions);
		thrust::transform(d_positions, d_positions + count, d_positions, InitializePositionsFunctor(modelMatrix));
	}

	__global__ void EstimatePositions_Impl(CONST(glm::vec3*) positions, glm::vec3* predicted, glm::vec3* velocities, float deltaTime)
	{
		GET_CUDA_ID(id, d_params.numParticles);

		glm::vec3 gravity = glm::vec3(0, -10, 0);
		velocities[id] += d_params.gravity * deltaTime;
		predicted[id] = positions[id] + velocities[id] * deltaTime;
	}

	void EstimatePositions(CONST(glm::vec3*) positions, glm::vec3* predicted, glm::vec3* velocities, float deltaTime)
	{
		CUDA_CALL(EstimatePositions_Impl, h_params.numParticles)(positions, predicted, velocities, deltaTime);
	}

	__device__ void AtomicAdd(glm::vec3* address, int index, glm::vec3 val)
	{
		atomicAdd(&(address[index].x), val.x);
		atomicAdd(&(address[index].y), val.y);
		atomicAdd(&(address[index].z), val.z);
	}

	__global__ void SolveStretch_Impl(uint numConstraints, CONST(int*) stretchIndices, CONST(float*) stretchLengths, 
		CONST(float*) inverseMass, CONST(glm::vec3*) predicted, glm::vec3* positionDeltas, int* positionDeltaCount)
	{
		GET_CUDA_ID(id, numConstraints);

		int idx1 = stretchIndices[2 * id];
		int idx2 = stretchIndices[2 * id + 1];
		float expectedDistance = stretchLengths[id];

		glm::vec3 diff = predicted[idx1] - predicted[idx2];
		float distance = glm::length(diff);
		float w1 = inverseMass[idx1];
		float w2 = inverseMass[idx2];

		if (distance > expectedDistance && w1 + w2 > 0)
		{
			glm::vec3 gradient = diff / (distance + EPSILON);
			// compliance is zero, therefore XPBD=PBD
			float denom = w1 + w2;
			float lambda = (distance - expectedDistance) / denom;
			glm::vec3 correction1 = -w1 * lambda * gradient;
			glm::vec3 correction2 = w2 * lambda * gradient;
			AtomicAdd(positionDeltas, idx1, correction1);
			AtomicAdd(positionDeltas, idx2, correction2);
			atomicAdd(&positionDeltaCount[idx1], 1);
			atomicAdd(&positionDeltaCount[idx2], 1);
			//printf("correction[%d] = (%.2f,%.2f,%.2f)\n", idx1, correction1.x, correction1.y, correction1.z);
			//printf("correction[%d] = (%.2f,%.2f,%.2f)\n", idx2, correction2.x, correction2.y, correction2.z);
		}
	}

	__global__ void ApplyPositionDeltas_Impl(glm::vec3* predicted, glm::vec3* positionDeltas, int* positionDeltaCount)
	{
		GET_CUDA_ID(id, d_params.numParticles);

		float count = (float)positionDeltaCount[id];
		if (count > 0)
		{
			predicted[id] += positionDeltas[id] / count;
			positionDeltas[id] = glm::vec3(0);
			positionDeltaCount[id] = 0;
		}
	}

	void SolveStretch(uint numConstraints, CONST(int*) stretchIndices, CONST(float*) stretchLengths,
		CONST(float*) inverseMass, glm::vec3* predicted, glm::vec3* positionDeltas, int* positionDeltaCount)
	{
		CUDA_CALL(SolveStretch_Impl, numConstraints)(numConstraints, stretchIndices, stretchLengths, inverseMass, predicted, positionDeltas, positionDeltaCount);
		CUDA_CALL(ApplyPositionDeltas_Impl, h_params.numParticles)(predicted, positionDeltas, positionDeltaCount);
	}

	__global__ void UpdatePositionsAndVelocities_Impl(CONST(glm::vec3*) predicted, glm::vec3* velocities, glm::vec3* positions, float deltaTime)
	{
		GET_CUDA_ID(id, d_params.numParticles);

		velocities[id] = (predicted[id] - positions[id]) / deltaTime * (1 - d_params.damping * deltaTime);
		positions[id] = predicted[id];
	}

	void UpdatePositionsAndVelocities(CONST(glm::vec3*) predicted, glm::vec3* velocities, glm::vec3* positions, float deltaTime)
	{
		CUDA_CALL(UpdatePositionsAndVelocities_Impl, h_params.numParticles)(predicted, velocities, positions, deltaTime);
	}

	__global__ void SolveAttachment_Impl(int numConstraints, CONST(int*) attachIndices, CONST(glm::vec3*) attachPositions, glm::vec3* predicted)
	{
		GET_CUDA_ID(id, numConstraints);

		predicted[attachIndices[id]] = attachPositions[id];
	}

	void SolveAttachment(int numConstraints, CONST(int*) attachIndices, CONST(glm::vec3*) attachPositions, glm::vec3* predicted)
	{
		CUDA_CALL(SolveAttachment_Impl, numConstraints)(numConstraints, attachIndices, attachPositions, predicted);
	}

	__device__ glm::vec3 ComputeFriction(glm::vec3 correction, glm::vec3 relVel)
	{
		glm::vec3 friction = glm::vec3(0);
		float correctionLength = glm::length(correction);
		if (d_params.friction > 0 && correctionLength > 0)
		{
			glm::vec3 norm = correction / correctionLength;

			glm::vec3 tanVel = relVel - norm * glm::dot(relVel, norm);
			float tanLength = glm::length(tanVel);
			float maxTanLength = correctionLength * d_params.friction;

			friction = -tanVel * min(maxTanLength / tanLength, 1.0f);
		}
		return friction;
	}

	__global__ void SolveSDFCollision_Impl(const uint numColliders, CONST(SDFCollider*) colliders, CONST(glm::vec3*) positions, glm::vec3* predicted)
	{
		GET_CUDA_ID(id, d_params.numParticles);

		glm::vec3 force = glm::vec3(0);
		for (int i = 0; i < numColliders; i++)
		{
			auto collider = colliders[i];
			auto pos = predicted[id];
			glm::vec3 correction = collider.ComputeSDF(pos, d_params.collisionMargin);
			force += correction;

			glm::vec3 relVel = predicted[id] - positions[id];
			auto friction = ComputeFriction(correction, relVel);
			force += friction;
		}
		predicted[id] += force;
	}

	void SolveSDFCollision(const uint numColliders, CONST(SDFCollider*) colliders, CONST(glm::vec3*) positions, glm::vec3* predicted)
	{
		if (numColliders == 0) return;
		
		CUDA_CALL(SolveSDFCollision_Impl, h_params.numParticles)(numColliders, colliders, positions, predicted);
	}

	__global__ void ComputeTriangleNormals(uint numTriangles, CONST(glm::vec3*) positions, CONST(uint*) indices, glm::vec3* normals)
	{
		GET_CUDA_ID(id, numTriangles);
		uint idx1 = indices[id * 3];
		uint idx2 = indices[id * 3+1];
		uint idx3 = indices[id * 3+2];

		auto p1 = positions[idx1];
		auto p2 = positions[idx2];
		auto p3 = positions[idx3];

		auto normal = glm::cross(p2 - p1, p3 - p1);
		AtomicAdd(normals, idx1, normal);
		AtomicAdd(normals, idx2, normal);
		AtomicAdd(normals, idx3, normal);
	}

	__global__ void ComputeVertexNormals(glm::vec3* normals)
	{
		GET_CUDA_ID(id, d_params.numParticles);

		//normals[id] = glm::vec3(0,1,0);
		normals[id] = glm::normalize(normals[id]);
	}

	void ComputeNormal(uint numTriangles, CONST(glm::vec3*) positions, CONST(uint*) indices, glm::vec3* normals)
	{
		if (h_params.numParticles)
		{
			hipMemset(normals, 0, h_params.numParticles * sizeof(glm::vec3));
				
			CUDA_CALL(ComputeTriangleNormals, numTriangles)(numTriangles, positions, indices, normals);
			CUDA_CALL(ComputeVertexNormals, h_params.numParticles)(normals);
		}
	}

	__global__ void SolveParticleCollision_Impl(
		CONST(float*) inverseMass,
		CONST(uint*) neighbors,
		CONST(glm::vec3*) positions,
		CONST(glm::vec3*) predicted,
		glm::vec3* positionDeltas, 
		int* positionDeltaCount)
	{
		GET_CUDA_ID(id, d_params.numParticles);

		glm::vec3 positionDelta = glm::vec3(0);
		int deltaCount = 0;
		glm::vec3 myVelocity = (predicted[id] - positions[id]);

		for (int j = id * d_params.maxNumNeighbors; j < (id+1) * d_params.maxNumNeighbors; j++)
		{
			uint idx1 = id;
			uint idx2 = neighbors[j];
			if (idx1 == idx2) continue;
			if (idx2 > d_params.numParticles) break;

			float expectedDistance = d_params.particleDiameter;

			glm::vec3 diff = predicted[idx1] - predicted[idx2];
			float distance = glm::length(diff);
			auto w1 = inverseMass[idx1];
			auto w2 = inverseMass[idx2];

			if (distance < expectedDistance && w1 + w2 > 0)
			{
				auto gradient = diff / (distance + EPSILON);
				auto denom = w1 + w2;
				auto lambda = (distance - expectedDistance) / denom;
				auto common = lambda * gradient;

				positionDelta -= w1 * common;
				deltaCount += 1;

				glm::vec3 relativeVelocity = myVelocity - (predicted[idx2] - positions[idx2]);
				auto friction = ComputeFriction(common, relativeVelocity);
				positionDelta += w1 * friction;
			}
		}

		positionDeltas[id]  = positionDelta;
		positionDeltaCount[id] = deltaCount;
	}

	void SolveParticleCollision(
		CONST(float*) inverseMass,
		CONST(uint*) neighbors,
		CONST(glm::vec3*) positions,
		glm::vec3* predicted,
		glm::vec3* positionDeltas,
		int* positionDeltaCount)
	{
		CUDA_CALL(SolveParticleCollision_Impl, h_params.numParticles)(inverseMass, neighbors, positions, predicted, positionDeltas, positionDeltaCount);
		CUDA_CALL(ApplyPositionDeltas_Impl, h_params.numParticles)(predicted, positionDeltas, positionDeltaCount);
	}
}