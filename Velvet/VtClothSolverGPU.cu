#include "hip/hip_runtime.h"
#include "VtClothSolverGPU.cuh"

#include <tuple>
#include <fmt/format.h>

#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>

using namespace std;

#define EPSILON 1e-6f

namespace Velvet
{
	__device__ __constant__ SimulationParams d_params;

	void SetSimulationParams(SimulationParams* hostParams)
	{
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_params), hostParams, sizeof(SimulationParams)));
	}

	struct InitializePositionsFunctor
	{
		const glm::mat4 matrix;
		InitializePositionsFunctor(glm::mat4 _matrix) : matrix(_matrix) {}

		__host__ __device__
			glm::vec3 operator()(const glm::vec3 position) const {
			return glm::vec3(matrix * glm::vec4(position, 1));
		}
	};

	void InitializePositions(glm::vec3* positions, int count, glm::mat4 modelMatrix)
	{
		thrust::device_ptr<glm::vec3> d_positions(positions);
		thrust::transform(d_positions, d_positions + count, d_positions, InitializePositionsFunctor(modelMatrix));
	}

	__global__ void ApplyExternalForces_Impl(glm::vec3* positions, glm::vec3* velocities)
	{
		uint id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= d_params.numParticles) return;

		glm::vec3 gravity = glm::vec3(0, -10, 0);
		velocities[id] += d_params.gravity * d_params.deltaTime;
		positions[id] += velocities[id] * d_params.deltaTime;
	}

	const uint blockSize = 256;

	void ComputeGridSize(uint n, uint &numBlocks, uint &numThreads)
	{
		if (n == 0)
		{
			fmt::print("Error(Solver): numParticles is 0\n");
			numBlocks = 0;
			numThreads = 0;
			return;
		}
		numThreads = min(n, blockSize);
		numBlocks = (n % numThreads != 0) ? (n / numThreads + 1) : (n / numThreads);
	}

	void ApplyExternalForces(glm::vec3* positions, glm::vec3* velocities, uint numParticles)
	{
		uint numBlocks, numThreads;
		ComputeGridSize(numParticles, numBlocks, numThreads);
		ApplyExternalForces_Impl <<< numBlocks, numThreads >>> (positions, velocities);
	}

	__device__ void AtomicAdd(glm::vec3* address, int index, glm::vec3 val)
	{
		atomicAdd(&(address[index].x), val.x);
		atomicAdd(&(address[index].y), val.y);
		atomicAdd(&(address[index].z), val.z);
	}

	__global__ void SolveStretch_Impl(glm::vec3* predicted, int* stretchIndices, float* stretchLengths, float* inverseMass, uint numConstraints)
	{
		uint id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= numConstraints) return;

		int idx1 = stretchIndices[2 * id];
		int idx2 = stretchIndices[2 * id + 1];
		float expectedDistance = stretchLengths[id];

		glm::vec3 diff = predicted[idx1] - predicted[idx2];
		float distance = glm::length(diff);
		float w1 = inverseMass[idx1];
		float w2 = inverseMass[idx2];

		if (distance > expectedDistance && w1 + w2 > 0)
		{
			auto gradient = diff / (distance + EPSILON);
			// compliance is zero, therefore XPBD=PBD
			auto denom = w1 + w2;
			auto lambda = (distance - expectedDistance) / denom;
			auto correction1 = -w1 * lambda * gradient;
			auto correction2 = w2 * lambda * gradient;
			AtomicAdd(predicted, idx1, correction1);
			AtomicAdd(predicted, idx2, correction2);
		}
	}

	void SolveStretch(glm::vec3* predicted, int* stretchIndices, float* stretchLengths, float* inverseMass, uint numConstraints)
	{
		uint numBlocks, numThreads;
		ComputeGridSize(numConstraints, numBlocks, numThreads);
		SolveStretch_Impl <<< numBlocks, numThreads >>> (predicted, stretchIndices, stretchLengths, inverseMass, numConstraints);
	}
}